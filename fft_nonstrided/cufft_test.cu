#include "../common/common.cuh"

float get_bandwith_scale_factor() {
    return 2.0f;
}

void make_cufft_handle(hipfftHandle* plan, long long data_size, int fft_size) {
    const long long dim1 = fft_size;
    const long long dim0 = data_size / dim1;

    checkCuFFT(hipfftCreate(plan), "hipfftCreate");
    checkCuFFT(hipfftPlan1d(plan, dim1, HIPFFT_C2C, dim0), "plan");
}

void exec_cufft_batch(hipfftHandle plan, hipfftComplex* d_data, hipfftComplex* d_kernel, long long total_elems) {
    checkCuFFT(hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD), "exec");
}