#include "hip/hip_runtime.h"
#include "../common/common.cuh"

float get_bandwith_scale_factor() {
    return 11.0f;
}

void make_cufft_handle(hipfftHandle* plan, long long data_size, int fft_size) {
    const long long dim2 = fft_size;
    const long long dim1 = fft_size;
    const long long dim0 = data_size / (dim1 * dim2);

    int n[2] = { int(dim1), int(dim2) };
    int inembed[2] = { int(dim1), int(dim2) };
    int onembed[2] = { int(dim1), int(dim2) };
    int istride    = 1;
    int ostride    = 1;
    int idist      = int(dim1)* int(dim2);
    int odist      = int(dim1)* int(dim2);

    checkCuFFT(hipfftPlanMany(plan, 2, n,
                                  inembed,  istride, idist,
                                  onembed,  ostride, odist,
                                  HIPFFT_C2C, int(dim0)), "plan2d");
}

__global__ void convolve_arrays(hipfftComplex* data, hipfftComplex* kernel, long long total_elems) {
    long long i = blockIdx.x * 1LL * blockDim.x + threadIdx.x;
    if (i < total_elems) {
        const size_t idx_in_image = i;
        const hipfftComplex d = data[i];
        const hipfftComplex k = kernel[idx_in_image];

        const float real = d.x * k.x - d.y * k.y;
        const float imag = d.x * k.y + d.y * k.x;
        data[i] = make_float2(real, imag);
    }
}

void exec_cufft_batch(hipfftHandle plan, hipfftComplex* d_data, hipfftComplex* d_kernel, long long total_elems) {
    checkCuFFT(hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD), "warmup");
    convolve_arrays<<<(total_elems+255)/256,256>>>(d_data, d_kernel, total_elems);
    checkCuFFT(hipfftExecC2C(plan, d_data, d_data, HIPFFT_BACKWARD), "warmup");
}