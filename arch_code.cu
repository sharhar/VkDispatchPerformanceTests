/*

Simple CUDA program that prionts to stdout the compute capability of a given GPU
as two digits, e.g., "86" for compute capability 8.6.

*/
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

int main(int argc, char** argv) {
    int dev = (argc > 1) ? std::atoi(argv[1]) : 0;

    int count = 0;
    hipError_t e = hipGetDeviceCount(&count);
    if (e != hipSuccess || count == 0) {
        std::fprintf(stderr, "No CUDA devices found: %s\n", hipGetErrorString(e));
        return 1;
    }
    if (dev < 0 || dev >= count) {
        std::fprintf(stderr, "Invalid device index %d (0..%d)\n", dev, count - 1);
        return 1;
    }

    hipDeviceProp_t prop{};
    e = hipGetDeviceProperties(&prop, dev);
    if (e != hipSuccess) {
        std::fprintf(stderr, "hipGetDeviceProperties failed: %s\n", hipGetErrorString(e));
        return 1;
    }

    std::printf("%d%d\n", prop.major, prop.minor); // e.g., 86
    return 0;
}
