#include "hip/hip_runtime.h"
#include "../common/common.cuh"

float get_bandwith_scale_factor() {
    return 6.0f;
}

void make_cufft_handle(hipfftHandle* plan, long long data_size, int fft_size) {
    const long long dim1 = fft_size;
    const long long dim0 = data_size / dim1;

    //checkCuFFT(hipfftCreate(plan), "hipfftCreate");
    checkCuFFT(hipfftPlan1d(plan, dim1, HIPFFT_C2C, dim0), "plan");
}

__global__ void scale_kernel(hipfftComplex* data, float scale_factor, long long total_elems) {
    long long i = blockIdx.x * 1LL * blockDim.x + threadIdx.x;
    if (i < total_elems) {
        data[i].x *= scale_factor;
        data[i].y *= scale_factor;
    }
}

void exec_cufft_batch(hipfftHandle plan, hipfftComplex* d_data, hipfftComplex* d_kernel, long long total_elems) {
    checkCuFFT(hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD), "warmup");
    scale_kernel<<<(total_elems+255)/256,256>>>(d_data, 5.0, total_elems);
    checkCuFFT(hipfftExecC2C(plan, d_data, d_data, HIPFFT_BACKWARD), "warmup");
}